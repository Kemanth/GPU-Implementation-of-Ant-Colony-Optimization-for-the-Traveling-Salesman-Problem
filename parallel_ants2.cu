
#include <hip/hip_runtime.h>
#include<iostream>
#include<fstream>
#include<math.h>
#include<stdlib.h>
#include<hiprand/hiprand_kernel.h>

#define MAX_CITIES 48 
#define MAX_ANTS 48			
#define Q 100
#define ALPHA 1.0
#define BETA 5.0 
#define RHO 0.5 

using namespace std;

int n=0;
int NC = 0;
int t = 0;
struct cities
{
	int x,y;
};
int s;
struct ants{
	
	int curCity, nextCity;
	int visited[MAX_CITIES];
	int tabu[MAX_CITIES];
	float L;
};

cities city[MAX_CITIES];
float pheromone[MAX_CITIES][MAX_CITIES];
float dist[MAX_CITIES][MAX_CITIES];
ants ant[MAX_ANTS];
float best=(double)999999;
int bestIndex;
float delta[MAX_CITIES][MAX_CITIES];
float fitness[MAX_CITIES][MAX_CITIES];
hiprandState  state[MAX_ANTS];


__global__ void initialize(float *d_dist,float *d_pheromone,float *d_delta,cities *d_city,int n)
{	
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	if((row<n)&&(col<n)){
	
		d_dist[col + row * n] = 0.0f;
		d_pheromone[col + row * n] = 1.0 / n;
		d_delta[col + row * n] = 0.0f;
		if(row!=col)
		{
			d_dist[col + row * n]=sqrt(powf(abs(d_city[row].x-d_city[col].x),2)+powf(abs(d_city[row].y-d_city[col].y),2));
			
		}
	}
}

__global__ void setup_curand_states(hiprandState *state_d, unsigned long t){
	
	int id = threadIdx.x + blockIdx.x*blockDim.x;
	hiprand_init(t, id, 0, &state_d[id]);
}

__global__ void initTour(ants *d_ant,int n){
	//cout << "inside init tour" << endl;
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if(id<n){
		int j = id;
		d_ant[id].curCity = j;
		for(int i=0;i<n;i++)
		{
			d_ant[id].visited[i]=0;
		}
		d_ant[id].visited[j] = 1;
		d_ant[id].tabu[0] = j;
		d_ant[id].L = 0.0;
	}
}

__global__ void calcFitness(float *d_fitness, float *d_dist, float *pheromone, int n){
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if(row < n && col < n){
		int id = row * n + col;
		d_fitness[id] =  powf( pheromone[id], ALPHA) * powf( (1.0/ d_dist[id]), BETA);
	}
}

__device__ int selectNextCity(int k,int n,float *d_fitness,ants *d_ant,hiprandState *state_d)
{	//cout<<"next city"<<endl;
	int i = d_ant[k].curCity;
	int j;
	double prod=0.0;
	for(j=0;j<n;j++)
	{
		if(d_ant[k].visited[j]==0)
		{
			prod+= d_fitness[i*n+j];
		}
	}
	
	while(1)
	{
		j++;
		if(j >= n)
			j=0;
		if(d_ant[k].visited[j] == 0)
		{
			double p = d_fitness[i*n+j]/prod;
			double x = ((double)(hiprand(&state_d[k])% 1000000000000000000)/1000000000000000000); 
			
			if(x < p)
			{
				break;
			}
		}
	}
	
	return j;
}

__global__ void tourConstruction(ants *d_ant, float *d_dist, float *d_fitness,int n,hiprandState *state_d)
{	//cout<<"tourConstruc"<<endl;
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if(id < n){
		for(int s=1;s<n;s++)
		{	
		
			int j = selectNextCity(id, n, d_fitness,d_ant,state_d);	
			d_ant[id].nextCity = j;
			d_ant[id].visited[j]=1;
			d_ant[id].tabu[s] = j;			
			d_ant[id].L+=d_dist[d_ant[id].curCity * n + j];
			d_ant[id].curCity = j;
		}
	}
}
void wrapUpTour(){
	//cout<<"wrapup"<<endl;
	for(int k = 0; k < MAX_ANTS;k++){
		ant[k].L += dist[ant[k].curCity][ant[k].tabu[0]];
		ant[k].curCity = ant[k].tabu[0];
		
		if(best > ant[k].L){
			best = ant[k].L;
			bestIndex = k;
		}
	}
}
void updatePheromone(){
	//cout<<"update"<<endl;
	for(int i =0;i<n;i++)
	{
		for(int j=0;j<n;j++)
		{
			if(i!=j)
			{
				pheromone[i][j] *=( 1.0 - RHO);
				
				if(pheromone[i][j]<0.0)
				{
					pheromone[i][j] = (1.0/n);
				}
			}
			pheromone[i][j] += delta[i][j];
			delta[i][j] = 0;
		}
	}
	t += MAX_ANTS;
	NC += 1;
}
void emptyTabu(){
	cout<<"emptytabu"<<endl;
	for(int k = 0;k<MAX_ANTS;k++){
		for(int i = 0; i < MAX_CITIES;i++){
			ant[k].tabu[i] = 0;
			ant[k].visited[i] = 0;
			int first = ant[k].tabu[i];
			int second = ant[k].tabu[(i + 1) % MAX_CITIES];
			delta[first][second] += Q/ant[k].L;
		}
	}
}

int main(int argc, char *argv[])
{	if (argc > 1){
		cout << "Reading File "<< argv[1]<<endl;
	}
	else{
		cout << "Usage:progname inputFileName" <<endl;
		return 1;
	}
	ifstream in;
    	in.open(argv[1]);
	in>>n;
	cout<<n<<endl;
	int num;
	for(int i=0;i<n;i++)
	{
		in>>num;	
		in>>city[i].x;
		in>>city[i].y;
		cout<<city[i].x<<" "<<city[i].y<<" "<<endl;	
	}
	
	dim3 blockDim(32, 32, 1);
	dim3 gridDim((n - 1)/ 32 + 1, (n - 1)/ 32 + 1, 1 );
	float *d_dist,*d_pheromone,*d_delta,*d_fitness;
	ants *d_ant;
	cities *d_city;
	hiprandState  *state_d;
	hipMalloc((void**)&d_pheromone, sizeof(float) * n * n);
	hipMalloc((void**)&d_dist, sizeof(float) * n * n);
	hipMalloc((void**)&d_delta, sizeof(float) * n * n);
	hipMalloc((void**)&d_ant, sizeof(ants) * n);
	hipMalloc((void**)&d_city, sizeof(cities) * n);
	hipMalloc((void**)&d_fitness, sizeof(float) * n *n);
	hipMalloc( (void**) &state_d, sizeof(state));
	hipMemcpy(d_city,city,sizeof(cities) * n,hipMemcpyHostToDevice);
	time_t t; 
	time(&t);
	setup_curand_states <<< (n-1)/32+1,32 >>> (state_d, (unsigned long) t);
	initialize<<<gridDim, blockDim>>>(d_dist,d_pheromone,d_delta,d_city,n);
	hipMemcpy(dist,d_dist,sizeof(float) * n * n,hipMemcpyDeviceToHost);
	hipMemcpy(pheromone,d_pheromone,sizeof(float) * n * n,hipMemcpyDeviceToHost);
	hipMemcpy(delta,d_delta,sizeof(float) * n * n,hipMemcpyDeviceToHost);
	int MAX_TIME = 20;
	for(;;)
	{		
		initTour<<<(n-1)/32+1,32>>>(d_ant,n);
		calcFitness<<< gridDim, blockDim>>>(d_fitness, d_dist, d_pheromone, n);
		tourConstruction<<<(n-1)/32+1,32>>>(d_ant,d_dist,d_fitness,n,state_d);
		hipMemcpy(ant,d_ant,sizeof(ants) * n,hipMemcpyDeviceToHost);
		wrapUpTour();
		updatePheromone();
		if(NC < MAX_TIME){
			emptyTabu();
		}
		else{
			break;
		}
	}
	cout<<endl;
	for(int i=0;i<n;i++)
	{
		cout<<ant[bestIndex].tabu[i]<<" ";
	}
	cout<<endl;
	cout<<"\nSACO: Best tour = "<<best<<endl<<endl<<endl;
	return 0;
}

