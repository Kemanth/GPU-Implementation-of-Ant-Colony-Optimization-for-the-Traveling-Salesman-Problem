#include "hip/hip_runtime.h"
#include<iostream>
#include<fstream>
#include<math.h>
#include<stdlib.h>
#include<hiprand/hiprand_kernel.h>
#include<hiprand.h>

#define MAX_CITIES 318	
#define MAX_ANTS 318		
#define Q 100
#define ALPHA 1.0
#define BETA 5.0 
#define RHO 0.5 

using namespace std;

int n=0;
int NC = 0;
int t = 0;
struct cities
{
	int x,y;
};
int s;
struct ants{
	
	int curCity, nextCity;
	int visited[MAX_CITIES];
	int tabu[MAX_CITIES];
	float L;
};

cities city[MAX_CITIES];
float pheromone[MAX_CITIES][MAX_CITIES];
float dist[MAX_CITIES][MAX_CITIES];
ants ant[MAX_ANTS];
int best=999999;
int bestIndex;
float delta[MAX_CITIES][MAX_CITIES];
float fitness[MAX_CITIES][MAX_CITIES];
hiprandState  state[MAX_ANTS];


__global__ void initialize(float *d_dist,float *d_pheromone,float *d_delta,cities *d_city,int n)
{	
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	if((row<n)&&(col<n)){
	
		d_dist[col + row * n] = 0.0f;
		d_pheromone[col + row * n] = 1.0f / n;
		d_delta[col + row * n] = 0.0f;
		if(row!=col)
		{
			d_dist[col + row * n]=sqrt(powf(abs(d_city[row].x-d_city[col].x),2)+powf(abs(d_city[row].y-d_city[col].y),2));
			
		}
	}
}

__global__ void setup_curand_states(hiprandState *state_d,int t){
	
	int id = threadIdx.x + blockIdx.x*blockDim.x;
	hiprand_init(t, id, 0, &state_d[id]);
}

__device__ float generate(hiprandState* globalState, int ind){
    //int ind = threadIdx.x;
    hiprandState localState = globalState[ind];
    float RANDOM = hiprand_uniform( &localState );
    globalState[ind] = localState;
    return RANDOM;
}

__global__ void initTour(ants *d_ant,int n){
	//cout << "inside init tour" << endl;
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if(id<n){
		int j = id;
		d_ant[id].curCity = j;
		for(int i=0;i<n;i++)
		{
			d_ant[id].visited[i]=0;
		}
		d_ant[id].visited[j] = 1;
		d_ant[id].tabu[0] = j;
		d_ant[id].L = 0.0;
	}
}

__global__ void calcFitness(float *d_fitness, float *d_dist, float *pheromone, int n){
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	if(row < n && col < n){
		int id = row * n + col;
		d_fitness[id] =  powf( pheromone[id], ALPHA) * powf( (1.0/ d_dist[id]), BETA);
	}
}

__device__ int selectNextCity(int k,int n,float *d_fitness,ants *d_ant,hiprandState *state_d)
{	//cout<<"next city"<<endl;
	int i = d_ant[k].curCity;
	int j;
	double prod=0.0;
	for(j=0;j<n;j++)
	{
		if(d_ant[k].visited[j]==0)
		{
			prod+= d_fitness[i*n+j];
		}
	}
	
	while(1)
	{
		j++;
		if(j >= n)
			j=0;
		if(d_ant[k].visited[j] == 0)
		{
			float p = d_fitness[i*n+j]/prod;
			float x = (float)generate(state_d,i); 
			
			if(x < p)
			{
				break;
			}
		}
	}
	
	return j;
}

__global__ void tourConstruction(ants *d_ant, float *d_dist, float *d_fitness,int n,hiprandState *state_d)
{	//printf("tour contruction\n");
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if(id < n){
		for(int s=1;s<n;s++)
		{	
			int j = selectNextCity(id, n, d_fitness,d_ant,state_d);	
			d_ant[id].nextCity = j;
			d_ant[id].visited[j]=1;
			d_ant[id].tabu[s] = j;			
			d_ant[id].L+=d_dist[d_ant[id].curCity * n + j];
			d_ant[id].curCity = j;
		}
	}
}
__global__
void wrapUpTour(float *delta, ants *ant,float *dist, int *best, int *bestIndex){
	//printf("wrap tour\n");
	int k = threadIdx.x + blockIdx.x * blockDim.x;
	if(k < MAX_ANTS){
		ant[k].L += dist[ant[k].curCity * MAX_CITIES + ant[k].tabu[0]];
		ant[k].curCity = ant[k].tabu[0];
		
		int temp = *best;
		printf("before best %d\n", *best);
		atomicMin(best, ant[k].L);
		printf("after best %d\n", *best);
		if (*best!= temp){
			*bestIndex = k;
		}
		for(int i = 0; i < MAX_CITIES;i++){
			int first = ant[k].tabu[i];
			int second = ant[k].tabu[(i + 1) % MAX_CITIES];
			delta[first * MAX_CITIES + second] += Q/ant[k].L;
		}
	}
	
}
__global__ void updatePheromone(float *d_pheromone, float *d_delta, int n){

	//printf("inside update phero\n");
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if(id < n){
		for(int s=0;s<n;s++){
			if(id!=s)
			{
				d_pheromone[id*n+s] *=( 1.0 - RHO);
				
				if(d_pheromone[id*n+s]<0.0)
				{
					d_pheromone[id*n+s] = (1.0/n);
				}
			}
			d_pheromone[id*n+s] += d_delta[id*n+s];
			d_delta[id*n+s] = 0;	
		}
	}
}
__global__ void emptyTabu(ants *d_ant,float *d_delta,int n){
	
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	
	if(id < n){
		//printf("Empty Tabu\n");
		for(int s=0;s<n;s++){		
			d_ant[id].tabu[s] = 0;
			d_ant[id].visited[s] = 0;
		}	
	}
}

int main(int argc, char *argv[])
{	if (argc > 1){
		cout << "Reading File "<< argv[1]<<endl;
	}
	else{
		cout << "Usage:progname inputFileName" <<endl;
		return 1;
	}
	ifstream in;
    	in.open(argv[1]);
	in>>n;
	cout<<n<<endl;
	int num;
	for(int i=0;i<n;i++)
	{
		in>>num;	
		in>>city[i].x;
		in>>city[i].y;
		cout<<city[i].x<<" "<<city[i].y<<" "<<endl;	
	}
	
	dim3 blockDim(32, 32, 1);
	dim3 gridDim((n - 1)/ 32 + 1, (n - 1)/ 32 + 1, 1 );
	float *d_dist,*d_pheromone,*d_delta,*d_fitness;
	ants *d_ant;
	cities *d_city;
	hiprandState  *state_d;
	int *d_best, *d_bestIndex;
	hipMalloc((void**)&d_pheromone, sizeof(float) * n * n);
	hipMalloc((void**)&d_dist, sizeof(float) * n * n);
	hipMalloc((void**)&d_delta, sizeof(float) * n * n);
	hipMalloc((void**)&d_ant, sizeof(ants) * n);
	hipMalloc((void**)&d_city, sizeof(cities) * n);
	hipMalloc((void**)&d_fitness, sizeof(float) * n *n);
	hipMalloc( (void**) &state_d, sizeof(state));
	hipMalloc((void **)&d_best, sizeof(int));
	hipMalloc((void **)&d_bestIndex, sizeof(int));
	hipMemcpy(d_city,city,sizeof(cities) * n,hipMemcpyHostToDevice);
	srand(time(0));
        hipMemcpy(d_best, &best, sizeof(int), hipMemcpyHostToDevice);	
	int seed = rand();
	setup_curand_states <<< (n-1)/32+1,32 >>> (state_d,seed);
	initialize<<<gridDim, blockDim>>>(d_dist,d_pheromone,d_delta,d_city,n);
	hipMemcpy(dist,d_dist,sizeof(float) * n * n,hipMemcpyDeviceToHost);
	hipMemcpy(pheromone,d_pheromone,sizeof(float) * n * n,hipMemcpyDeviceToHost);
	hipMemcpy(delta,d_delta,sizeof(float) * n * n,hipMemcpyDeviceToHost);
	int MAX_TIME = 20;
	for(;;)
	{		
		initTour<<<(n-1)/32+1,32>>>(d_ant,n);
		hipDeviceSynchronize();
		calcFitness<<< gridDim, blockDim>>>(d_fitness, d_dist, d_pheromone, n);
		hipDeviceSynchronize();
		tourConstruction<<<(n-1)/32+1,32>>>(d_ant,d_dist,d_fitness,n,state_d);
		hipDeviceSynchronize();
		hipMemcpy(ant,d_ant,sizeof(ants) * n,hipMemcpyDeviceToHost);
		wrapUpTour<<<(n - 1)/32 + 1, 32>>>(d_delta, d_ant, d_dist, d_best, d_bestIndex);
		updatePheromone<<< (n-1)/32+1,32>>>(d_pheromone,d_delta,n);
		hipDeviceSynchronize();
		t += MAX_ANTS;
		NC += 1;
		if(NC < MAX_TIME){
			emptyTabu<<<(n-1)/32+1,32>>>(d_ant,d_delta,n);
			hipMemcpy(&best, d_best, sizeof(int), hipMemcpyDeviceToHost);
			cout<<"Best Tour so far -->  "<<best<<endl;
			hipDeviceSynchronize();
		}
		else{
			break;
		}
	}
	cout<<endl;
	hipMemcpy(&best, d_best, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&bestIndex, d_bestIndex, sizeof(int), hipMemcpyDeviceToHost);

	for(int i=0;i<n;i++)
	{
		cout<<ant[bestIndex].tabu[i]<<" ";
	}
	cout<<endl;
	cout<<"\nSACO: Best tour = "<<best<<endl<<endl<<endl;
	return 0;
}

