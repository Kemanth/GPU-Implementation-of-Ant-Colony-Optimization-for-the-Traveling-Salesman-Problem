#include<iostream>
#include<fstream>
#include<math.h>
#include<stdlib.h>
#include <hip/hip_runtime.h>

#include <hiprand/hiprand_kernel.h>
#include <math.h>

#define MAX_CITIES 5000
#define MAX_ANTS 5000
#define QVAL 100
#define ALPHA 1.0
#define BETA 5.0 
#define RHO 0.5 

using namespace std;

struct cities
{
	int x,y;
};
struct ants{
	
	int curCity, nextCity, pathIndex;
	int visited[MAX_CITIES];
	int path[MAX_CITIES];
	double tourLength;
};

int n=0;
cities city[MAX_CITIES];
float pheromone[MAX_CITIES][MAX_CITIES];
float dist[MAX_CITIES][MAX_CITIES];
ants ant[MAX_ANTS];
float best=(double)999999;
int bestIndex;

__global__ void initialize(float *dist, float *pheromone, ants *ant,cities *city,int n){
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	if((row<n)&&(col<n)){
	
		dist[col + row * n] = 0.0f;
		pheromone[col + row * n] = 1.0 / n;
		if(row!=col)
		{
			dist[col + row * n]=sqrt(powf(abs(city[row].x-city[col].x),2)+powf(abs(city[row].y-city[col].y),2));
			dist[col + row * n]=dist[col + row * n];
		}
		ant[row].visited[col]=0;
		ant[row].path[col]=-1;
		if(col==0){
			ant[row].curCity=row;
			ant[row].pathIndex = 1;
			ant[row].path[0] = ant[row].curCity;
			ant[row].nextCity = -1;
			ant[row].tourLength = 0;
			ant[row].visited[ant[row].curCity]=1;
		}
	}
}

double fitness(int i, int j)
{
	return(( pow( pheromone[i][j], ALPHA) * pow( (1.0/ dist[i][j]), BETA)));
}

int selectNextCity(int k,int n)
{
	int i = ant[k].curCity;
	int j;
	double prod=0.0;
	for(j=0;j<n;j++)
	{
		if(ant[k].visited[j]==0)
		{
			prod+= fitness(i,j);
		}
	}
	
	while(1)
	{
		j++;
		
		if(j >= n)
			j=0;
		if(ant[k].visited[j] == 0)
		{
			double p = fitness(i,j)/prod;
			double x = ((double)rand()/RAND_MAX); 
			
			if(x < p)
			{
				break;
			}
		}
	}
	
	return j;
}

int tourConstruction()
{
	int movement=0;
	
	for(int i=0;i<n;i++)
	{
		if(ant[i].pathIndex < n)
		{
			ant[i].nextCity = selectNextCity(i,n);
			ant[i].visited[ant[i].nextCity]=1;
			ant[i].path[ant[i].pathIndex]=ant[i].nextCity;
			ant[i].pathIndex++;
			ant[i].tourLength+=dist[ant[i].curCity][ant[i].nextCity];
			
			if(ant[i].pathIndex == n)
			{//changed here
				ant[i].tourLength+=dist[ant[i].path[n-1]][ant[i].path[0]];
			}
			ant[i].curCity = ant[i].nextCity;
			movement++;
		}
	}
	return movement;
}

int updatePheromones()
{
	for(int i=0;i<n;i++)
	{
		for(int j=0;j<n;j++)
		{
			if(i!=j)
			{
				pheromone[i][j] *=( 1.0 - RHO);
				
				if(pheromone[i][j]<0.0)
				{
					pheromone[i][j] = (1.0/n);
				}
			}
		}
	}
	int a,b;
	for(int i=0;i<n;i++)
	{
		for(int j=0;j<n;j++)
		{
			if(j==n-1)
			{
				a=ant[i].path[j];
				b=ant[i].path[j+1];
			}
			else
			{
				a=ant[i].path[j];
				b=ant[i].path[0];
			}
			
			pheromone[a][b]+=(QVAL)/ant[i].tourLength;
			pheromone[b][a]+=pheromone[a][b];
		}
	}
}

void reDeployAnts()
{
	int visit=0;
	for(int i=0;i<n;i++)
	{
		if(ant[i].tourLength < best)
		{
			best = ant[i].tourLength;
			bestIndex = i;
			break;
		}
		if(i==n)
			visit=0;
		ant[i].curCity=visit++;
		for(int j=0;j<n;j++)
		{
			ant[i].visited[j]=0;
			ant[i].path[j]=-1;
		}
		ant[i].pathIndex = 1;
		ant[i].path[0] = ant[i].curCity;
		ant[i].nextCity = -1;
		ant[i].tourLength = 0;
		ant[i].visited[ant[i].curCity]=1;
	}
}

int main(int argc, char *argv[])
{	if (argc > 1){
		cout << "Reading File "<< argv[1]<<endl;
	}
	else{
		cout << "Usage:Progname inputFileName" << endl;
		return 1;
	}
	ifstream in;
    	in.open(argv[1]);
	in>>n;
	cout<<n<<endl;
	int num;
	for(int i=0;i<n;i++)
	{
		in>>num;	
		in>>city[i].x;
		in>>city[i].y;
		cout<<city[i].x<<" "<<city[i].y<<" "<<endl;	
	}
	dim3 blockDim(1024, 1024, 1);
	dim3 gridDim((n - 1)/ 1024 + 1, (n - 1)/ 1024 + 1, 1 );
	float *d_dist,*d_pheromone;
	ants *d_ant;
	cities *d_city;
	hipMalloc((void**)&d_pheromone, sizeof(float) * n * n);
	hipMalloc((void**)&d_dist, sizeof(float) * n * n);
	hipMalloc((void**)&d_ant, sizeof(ants) * n);
	hipMalloc((void**)&d_city, sizeof(cities) * n);
	hipMemcpy(d_city,city,sizeof(cities) * n,hipMemcpyHostToDevice);
	initialize<<<gridDim,blockDim>>>(d_dist,d_pheromone,d_ant,d_city,n);
	
	/*
	initialize();
	int MAX_TIME = 20 * n;
	for(int i=0;i<MAX_TIME;i++)
	{
		if( tourConstruction() == 0)
		{
			updatePheromones();
			
			if(i != MAX_TIME)
				reDeployAnts();
				
			cout<<"\n Time is "<<i<<"("<<best<<")";
			
		}
	}
	cout<<endl;
	for(int i=0;i<n;i++)
	{
		cout<<ant[bestIndex].path[i]<<" ";
	}
	cout<<endl;
	cout<<"\nSACO: Best tour = "<<best<<endl<<endl<<endl;
	*/
	return 0;
}

